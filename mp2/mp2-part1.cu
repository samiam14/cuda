#include "hip/hip_runtime.h"
/* This is machine problem 2, binning
 * The problem is that you have particles in a 3D domain
 * which is quantized into blocks or bins. You want to figure
 * out which block each particle belongs to.
 * Use the atomic functions that you learned about in lecture 3
 * to implement the same functionality as the reference version on the cpu.
 *
 * FOR EXTRA CREDIT: 
 * Write a version of your binning kernel that uses atomics hierarchically, 
 * accumulating updates first into shared memory and then merging the results 
 * from shared memory into the global memory. 
 * As a hint, think about binning particles first into a coarse grid in a first kernel,
 * and then binning the particles from each coarse bin into the 
 * final bins in a second kernel.
 */



/*
 * SUBMISSION INSTRUCTIONS
 * =========================
 * 
 * You can submit your entire working directory for this assignment 
 * from any of the cluster machines by using our submit script. We want to be able
 * to just run "make" to compile your code.
 * The submit script bundles the entire current directory into
 * a submission. Thus, you use it by CDing to a the directory for your assignment,
 * and running:
 * 
 *   > cd *some directory*
 *   > /usr/class/cs193g/bin/submit mp2
 * 
 * This will submit the current directory as your assignment. You can submit
 * as many times as you want, and we will use your last submission.
 */

#include <stdlib.h>
#include <stdio.h>
#include <ctime>

#include <hip/hip_runtime.h>

#include "mp2-util.h"

// TODO enable this to print debugging information
//const bool print_debug = true;
const bool print_debug = false;

event_pair timer;

// the particle coordinates are already normalized (in the domain [0,1] )
// gridding provides the base 2 log of how finely the domain is subdivided
// in each direction. So gridding.x == 6 means that the x-axis is subdivided
// into 64 parts. (i.e. 2^(gridding.x) = number of bins on x axis)
// Overall there cannot be more than 4B bins, so we can just concatenate the bin
// indices into a single uint.

__host__ __device__ unsigned int bin_index(float3 particle, int3 gridding) 
{
  unsigned int x_index = (unsigned int)(particle.x * (1 << gridding.x));
  unsigned int y_index = (unsigned int)(particle.y * (1 << gridding.y));
  unsigned int z_index = (unsigned int)(particle.z * (1 << gridding.z));
  unsigned int index = 0;
  index |= z_index;
  index <<= gridding.y;
  index |= y_index;
  index <<= gridding.x;
  index |=  x_index;

  return index;
}

void host_binning(float3 *particles, int *bins, int *bin_counters, int *overflow_flag, int3 gridding, int bin_size, int array_length)
{
  for(int i=0;i<array_length;i++)
  {
    unsigned int bin = bin_index(particles[i],gridding);
    if(bin_counters[bin] < bin_size)
    {
      unsigned int offset = bin_counters[bin];
      // let's not do the whole precrement / postcrement thing...
      bin_counters[bin]++;
      bins[bin*bin_size + offset] = i;
    }
    else {
      *overflow_flag = true;
    }

  }
}

bool cross_check_results(int * h_bins, int * h_bins_checker, int * h_bin_counters, int * h_bin_counters_checker, int * h_particles_binids_checker, int num_particles, int num_bins, int bin_size)
{
  int error = 0;

  for(int i=0;i<num_bins;i++)
  {
    if(h_bin_counters[i] != h_bin_counters_checker[i])
    {

      if(print_debug) fprintf(stderr,"mismatch! bin %d: cuda:%d host:%d particles \n",i,h_bin_counters[i],h_bin_counters_checker[i]);
      error = 1;
    }
    for(int j=0; j<bin_size;j++)
    {
      // record which these particles went into bin i in the reference version
      if(h_bins_checker[i*bin_size+j] != -1)
      {
        h_particles_binids_checker[h_bins_checker[i*bin_size+j]] = i;
      }
    }
    for(int j=0; j<bin_size;j++)
    {
      if(h_bins_checker[i*bin_size+j] != -1)
      {
        if(h_particles_binids_checker[h_bins[i*bin_size+j]] != i)
        {
          error = 1;
        }
      }
    }
  }

  if(error)
  {
    printf("Output of CUDA version and normal version didn't match! \n");
  }
  else {
    printf("Worked! CUDA and reference output match. \n");
  }
  return error;
}

template
<typename T>
__global__ void initialize(T *array,T value, unsigned int array_length)
{
  int gid = blockIdx.x * blockDim.x  + threadIdx.x;
  
  if(gid < array_length)
  {
    array[gid] = value;
  }
}

__global__ void particle_binning(float3 *particles, int *bins, unsigned int *bin_counters, int3 gridding, int num_particles, int bin_size)
{
	// Global id and bounds checking
	int gid = blockIdx.x * blockDim.x + threadIdx.x;
	if(gid >= num_particles) {
		return;
	}
	
	unsigned int bin = bin_index(particles[gid], gridding);
	unsigned int offset = atomicInc(&bin_counters[bin], bin_size); // Should check for overflow, but the host function does that already
	bins[bin * bin_size + offset] = gid;
}

void device_binning(float3 * h_particles, int * h_bins, int * h_bin_counters, int3 gridding, int num_particles, int num_bins, unsigned int bin_size)
{
	// Device pointers
	float3 *d_particles = 0;
	int *d_bins = 0;
	unsigned int *d_bin_counters = 0;
	
	// Cuda memory allocation
	hipMalloc((void**)&d_particles, num_particles * sizeof(float3));
	hipMalloc((void**)&d_bins, num_bins * bin_size * sizeof(unsigned int));
	hipMalloc((void**)&d_bin_counters, num_bins * sizeof(unsigned int));
	
	if(d_particles == 0 || d_bins == 0 || d_bin_counters == 0) {
		printf("error allocating memory");
		exit(1);
	}
	
	// Cuda memory copy (host to device)
	hipMemcpy(d_particles, h_particles, num_particles * sizeof(float3), hipMemcpyHostToDevice);
	
	// Grid dimensions
	int block_size = 512;
	int num_blocks_counters = (num_bins + block_size - 1) / block_size;
	int num_blocks_bins = (num_bins * bin_size + block_size - 1) / block_size;
	int num_blocks_particles = (num_particles + block_size - 1) / block_size;
	
	// Initialize the counters (shpould be done with hipMemset, but use the templated function since its available)
	initialize<<< num_blocks_counters, block_size >>>(d_bin_counters, (unsigned int)0, num_bins);
	initialize<<< num_blocks_bins, block_size >>>(d_bins, -1, num_bins * bin_size);
	
	start_timer(&timer);
	
	// Do the binning
	particle_binning<<< num_blocks_particles, block_size >>>(d_particles, d_bins, d_bin_counters, gridding, num_particles, bin_size);
	
	stop_timer(&timer, "gpu binning");
	
	// Cuda memory copy (device to host)
	hipMemcpy(h_bins, d_bins, num_bins * bin_size * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(h_bin_counters, d_bin_counters, num_bins * sizeof(int), hipMemcpyDeviceToHost);

	// Cuda deallocation
	hipFree(d_particles);
	hipFree(d_bins);
	hipFree(d_bin_counters);
}

int main(void)
{  
  // create arrays of 2M elements
  int num_particles = 1<<22;
  int log_bpd = 6;
  int bins_per_dim = 1 << log_bpd;
  unsigned int num_bins = bins_per_dim*bins_per_dim*bins_per_dim;
  // extra space to account for load imbalance to prevent frequent aborts due to bin overflow 
  int bin_size = num_particles/num_bins * 3;
  int3 gridding = make_int3(log_bpd,log_bpd,log_bpd);
  
  float3 *h_particles = 0;
  int *h_bins = 0;
  int *h_bin_counters = 0;
  int *h_bins_checker = 0;
  float3 *h_particles_checker = 0;
  int *h_bin_counters_checker = 0;
  int *h_particles_binids_checker = 0;
  int h_overflow_flag_checker = 0;

  // malloc host array
  h_particles = (float3*)malloc(num_particles * sizeof(float3));
  h_bins = (int*)malloc(num_bins * bin_size * sizeof(int));
  h_bin_counters = (int*)malloc(num_bins * sizeof(int));
  h_particles_checker = (float3*)malloc(num_particles * sizeof(float3));
  h_bins_checker = (int*)malloc(num_bins * bin_size * sizeof(int));
  h_particles_binids_checker = (int*)malloc(num_bins * bin_size * sizeof(int));
  h_bin_counters_checker = (int*)malloc(num_bins * sizeof(int));

  // if either memory allocation failed, report an error message
  if(h_particles == 0 ||  
      h_bins == 0 || h_bin_counters == 0 ||  
      h_bins_checker == 0 || h_bin_counters_checker == 0 ||
      h_particles_binids_checker == 0)
  {
    printf("couldn't allocate memory\n");
    exit(1);
  }
  
  // generate random input
  // initialize
  srand(13);

  for(int i=0;i< num_particles;i++)
  {
    h_particles[i] = h_particles_checker[i] = make_float3((float)rand()/(float)RAND_MAX,(float)rand()/(float)RAND_MAX,(float)rand()/(float)RAND_MAX);
  }
  for(int i=0;i<num_bins;i++)
  {
    h_bin_counters[i] = h_bin_counters_checker[i] = 0;
  }
  for(int i=0;i<num_bins*bin_size;i++)
  {
    h_bins[i] = h_bins_checker[i] = h_particles_binids_checker[i] = -1;
  }

  device_binning(h_particles, h_bins, h_bin_counters, gridding, num_particles, num_bins, bin_size);
  
  // generate reference output
  start_timer(&timer);
  host_binning(h_particles_checker, h_bins_checker, h_bin_counters_checker, &h_overflow_flag_checker, gridding, bin_size, num_particles);
  stop_timer(&timer,"cpu binning");
  
  if(h_overflow_flag_checker)
  {
    printf("one of the bins overflowed!\n");
    exit(1);
  }

  // check CUDA output versus reference output
  cross_check_results(h_bins, h_bins_checker, h_bin_counters, h_bin_counters_checker, h_particles_binids_checker, num_particles, num_bins, bin_size);

  // deallocate memory
  free(h_particles);
  free(h_bins);
  free(h_bin_counters);
  free(h_particles_checker);
  free(h_bins_checker);
  free(h_particles_binids_checker);
  free(h_bin_counters_checker);
 
  return 0;
}

