#include "hip/hip_runtime.h"
// This is machine problem 1, part 3, page ranking
// The problem is to compute the rank of a set of webpages
// given a link graph, aka a graph where each node is a webpage,
// and each edge is a link from one page to another.
// We're going to use the Pagerank algorithm (http://en.wikipedia.org/wiki/Pagerank),
// specifically the iterative algorithm for calculating the rank of a page
// We're going to run 20 iterations of the propage step.
// Implement the corresponding code in CUDA.

/* SUBMISSION GUIDELINES:
 * You should copy your entire device_graph_iterate fuction and the 
 * supporting kernal into a file called mp1-part3-solution.cu and submit
 * that file. The fuction needs to have the exact same interface as the 
 * device_graph_iterate function we provided. The kernel is internal 
 * to your code and can look any way you want.
 */


#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <ctime>
#include <limits>

#include "mp1-util.h"

event_pair timer;

// amount of floating point numbers between answer and computed value 
// for the answer to be taken correctly. 2's complement magick.
const int maxUlps = 1000;
  
void host_graph_propagate(unsigned int *graph_indices, unsigned int *graph_edges, float *graph_nodes_in, float *graph_nodes_out, float * inv_edges_per_node, int array_length)
{
  for(int i=0; i < array_length; i++)
  {
    float sum = 0.f; 
    for(int j = graph_indices[i]; j < graph_indices[i+1]; j++)
    {
      sum += graph_nodes_in[graph_edges[j]]*inv_edges_per_node[graph_edges[j]];
    }
    graph_nodes_out[i] = 0.5f/(float)array_length + 0.5f*sum;
  }
}


void host_graph_iterate(unsigned int *graph_indices, unsigned int *graph_edges, float *graph_nodes_A, float *graph_nodes_B, float * inv_edges_per_node, int nr_iterations, int array_length)
{
  assert((nr_iterations % 2) == 0);
  for(int iter = 0; iter < nr_iterations; iter+=2)
  {
    host_graph_propagate(graph_indices, graph_edges, graph_nodes_A, graph_nodes_B, inv_edges_per_node, array_length);
    host_graph_propagate(graph_indices, graph_edges, graph_nodes_B, graph_nodes_A, inv_edges_per_node, array_length);
  }
}


__global__ void device_graph_propogate(unsigned int *graph_indices,
										unsigned int *graph_edges,
										float *graph_nodes_in,
										float *graph_nodes_out,
										float *inv_edges_per_node,
										int array_length)
{
	// Global thread index and bounds checking
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i >= array_length)
		return;
	
	float sum = 0.f; 
	for(int j = graph_indices[i]; j < graph_indices[i+1]; j++)
	{
		sum += graph_nodes_in[graph_edges[j]] * inv_edges_per_node[graph_edges[j]];
	}
	graph_nodes_out[i] = 0.5f / (float)array_length + 0.5f * sum;
}


void device_graph_iterate(unsigned int *h_graph_indices,
                          unsigned int *h_graph_edges,
                          float *h_graph_nodes_input,
                          float *h_graph_nodes_result,
                          float *h_inv_edges_per_node,
                          int nr_iterations,
                          int num_elements,
                          int avg_edges)
{
	// Device pointers
	unsigned int *d_graph_indices = 0;
	unsigned int *d_graph_edges = 0;
	float *d_graph_nodes_input = 0;
	float *d_graph_nodes_result = 0;
	float *d_inv_edges_per_node = 0;
	
	// Cuda memory allocation
	hipMalloc((void**)&d_graph_indices, (num_elements + 1) * sizeof(unsigned int));
	hipMalloc((void**)&d_graph_edges, num_elements * avg_edges * sizeof(unsigned int));
	hipMalloc((void**)&d_graph_nodes_input, num_elements * sizeof(float));
	hipMalloc((void**)&d_graph_nodes_result, num_elements * sizeof(float));
	hipMalloc((void**)&d_inv_edges_per_node, num_elements * sizeof(float));
	
	if(d_graph_indices == 0 || d_graph_edges == 0 || d_graph_nodes_input == 0 || d_graph_nodes_result == 0 || d_inv_edges_per_node == 0) {
		printf("error allocating memory");
		exit(1);
	}
	
	// Cuda memory copy (host to device)
	hipMemcpy(d_graph_indices, h_graph_indices, (num_elements + 1) * sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(d_graph_edges, h_graph_edges, num_elements * avg_edges * sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(d_graph_nodes_input, h_graph_nodes_input, num_elements * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_graph_nodes_result, h_graph_nodes_result, num_elements * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_inv_edges_per_node, h_inv_edges_per_node, num_elements * sizeof(float), hipMemcpyHostToDevice);
	
	start_timer(&timer);
	
	int block_size = 512;
	int num_blocks = (num_elements + block_size - 1) / block_size;
	
	assert((nr_iterations % 2) == 0);
	for(int i = 0; i < nr_iterations; i += 2) {
		device_graph_propogate<<< num_blocks, block_size >>>(d_graph_indices, d_graph_edges, d_graph_nodes_input, d_graph_nodes_result, d_inv_edges_per_node, num_elements);
		device_graph_propogate<<< num_blocks, block_size >>>(d_graph_indices, d_graph_edges, d_graph_nodes_result, d_graph_nodes_input, d_inv_edges_per_node, num_elements);
	}
	
	check_launch("gpu graph propagate");
	stop_timer(&timer,"gpu graph propagate");
	
	// Cuda memory copy (device to host)
	hipMemcpy(h_graph_nodes_result, d_graph_nodes_result, num_elements * sizeof(float), hipMemcpyDeviceToHost);
	
	// Cuda memory free
	hipFree(d_graph_indices);
	hipFree(d_graph_edges);
	hipFree(d_graph_nodes_input);
	hipFree(d_graph_nodes_result);
	hipFree(d_inv_edges_per_node);
}


int main(void)
{
  // create arrays of 2M elements
  int num_elements = 1 << 21;
  int avg_edges = 8;
  int iterations = 20;
  
  // pointers to host & device arrays
  unsigned int *h_graph_indices = 0;
  float *h_inv_edges_per_node = 0;
  unsigned int *h_graph_edges = 0;
  float *h_graph_nodes_input = 0;
  float *h_graph_nodes_result = 0;
  float *h_graph_nodes_checker_A = 0;
  float *h_graph_nodes_checker_B = 0;
  
  
  // malloc host array
  // index array has to be n+1 so that the last thread can 
  // still look at its neighbor for a stopping point
  h_graph_indices = (unsigned int*)malloc((num_elements+1) * sizeof(unsigned int));
  h_inv_edges_per_node = (float*)malloc((num_elements) * sizeof(float));
  h_graph_edges = (unsigned int*)malloc(num_elements * avg_edges * sizeof(unsigned int));
  h_graph_nodes_input = (float*)malloc(num_elements * sizeof(float));
  h_graph_nodes_result = (float*)malloc(num_elements * sizeof(float));
  h_graph_nodes_checker_A = (float*)malloc(num_elements * sizeof(float));
  h_graph_nodes_checker_B = (float*)malloc(num_elements * sizeof(float));
  
  // if any memory allocation failed, report an error message
  if(h_graph_indices == 0 || h_graph_edges == 0 || h_graph_nodes_input == 0 || h_graph_nodes_result == 0 || 
	 h_inv_edges_per_node == 0 || h_graph_nodes_checker_A == 0 || h_graph_nodes_checker_B == 0)
  {
    printf("couldn't allocate memory\n");
    exit(1);
  }

  // generate random input
  // initialize
  srand(time(NULL));
   
  h_graph_indices[0] = 0;
  for(int i=0;i< num_elements;i++)
  {
    int nr_edges = (i % 15) + 1;
    h_inv_edges_per_node[i] = 1.f/(float)nr_edges;
    h_graph_indices[i+1] = h_graph_indices[i] + nr_edges;
    if(h_graph_indices[i+1] >= (num_elements * avg_edges))
    {
      printf("more edges than we have space for\n");
      exit(1);
    }
    for(int j=h_graph_indices[i];j<h_graph_indices[i+1];j++)
    {
      h_graph_edges[j] = rand() % num_elements;
    }
    
    h_graph_nodes_input[i] =  1.f/(float)num_elements;
    h_graph_nodes_checker_A[i] =  h_graph_nodes_input[i];
    h_graph_nodes_result[i] = std::numeric_limits<float>::infinity();
  }
  
  device_graph_iterate(h_graph_indices, h_graph_edges, h_graph_nodes_input, h_graph_nodes_result, h_inv_edges_per_node, iterations, num_elements, avg_edges);
  
  start_timer(&timer);
  // generate reference output
  host_graph_iterate(h_graph_indices, h_graph_edges, h_graph_nodes_checker_A, h_graph_nodes_checker_B, h_inv_edges_per_node, iterations, num_elements);
  
  check_launch("host graph propagate");
  stop_timer(&timer,"host graph propagate");
  
  // check CUDA output versus reference output
  int error = 0;
  int num_errors = 0;
  for(int i=0;i<num_elements;i++)
  {
    float n = h_graph_nodes_result[i];
    float c = h_graph_nodes_checker_A[i];
    if(!AlmostEqual2sComplement(n,c,maxUlps)) 
    {
      num_errors++;
      if (num_errors < 10)
      {
            printf("%d:%.3f::",i, n-c);
      }
      error = 1;
    }
  }
  
  if(error)
  {
    printf("Output of CUDA version and normal version didn't match! \n");
  }
  else
  {
    printf("Worked! CUDA and reference output match. \n");
  }

  // deallocate memory
  free(h_graph_indices);
  free(h_inv_edges_per_node);
  free(h_graph_edges);
  free(h_graph_nodes_input);
  free(h_graph_nodes_result);
  free(h_graph_nodes_checker_A);
  free(h_graph_nodes_checker_B);
}

