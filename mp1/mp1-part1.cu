#include "hip/hip_runtime.h"
/* This is machine problem 1, part 1, shift cypher
 *
 * The problem is to take in a string of unsigned ints and an int,
 * the shift amount, and add the number to each element of
 * the string, effectively "shifting" each element in the 
 * string.
 * SUBMISSION GUIDELINES:
 * You should copy the complete shift_cyper function from your solution
 * into a file called mp1-part1-solution-kernel.cu and submit that file.
 * The function needs to have exactly the same interface (including __global__)
 * as the empty shift_cypher function given below. 
 */


#include <stdlib.h>
#include <stdio.h>
#include <ctime>

#include "mp1-util.h"


// Repeating from the tutorial, just in case you haven't looked at it.

// "kernels" or __global__ functions are the entry points to code that executes on the GPU
// The keyword __global__ indicates to the compiler that this function is a GPU entry point.
// __global__ functions must return void, and may only be called or "launched" from code that
// executes on the CPU.

void host_shift_cypher(unsigned int *input_array, unsigned int *output_array, unsigned int shift_amount, unsigned int alphabet_max, unsigned int array_length)
{
  for(unsigned int i=0;i<array_length;i++)
  {
    int element = input_array[i];
    int shifted = element + shift_amount;
    if(shifted > alphabet_max)
    {
      shifted = shifted % (alphabet_max + 1);
    }
    output_array[i] = shifted;
  }
}


// This kernel implements a per element shift
__global__ void shift_cypher(unsigned int *input_array, unsigned int *output_array, unsigned int shift_amount, unsigned int alphabet_max, unsigned int array_length)
{
	// Calculate global thread index and check bounds
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i >= array_length)
		return;
	
	// Copied from above (host_shift_cypher)
	int element = input_array[i];
	int shifted = element + shift_amount;
	if(shifted > alphabet_max)
	{
	  shifted = shifted % (alphabet_max + 1);
	}
	output_array[i] = shifted;
}


int main(void)
{
  // initialize
  srand(time(NULL));
  
  // create arrays of 16M elements
  int num_elements = 1 << 24;

  
  unsigned int alphabet_max = 45647;
  
  // compute the size of the arrays in bytes
  int num_bytes = num_elements * sizeof(unsigned int);

  // pointers to host & device arrays
  unsigned int *host_input_array = 0;
  unsigned int *host_output_array = 0;
  unsigned int *host_output_checker_array = 0;
  unsigned int *device_input_array = 0;
  unsigned int *device_output_array = 0;
  
  event_pair timer;
  

  // malloc host arrays
  host_input_array = (unsigned int*)malloc(num_bytes);
  host_output_array = (unsigned int*)malloc(num_bytes);
  host_output_checker_array = (unsigned int*)malloc(num_bytes);

  // hipMalloc device arrays
  hipMalloc((void**)&device_input_array, num_bytes);
  hipMalloc((void**)&device_output_array, num_bytes);
  
  // if either memory allocation failed, report an error message
  if(host_input_array == 0 || host_output_array == 0 || host_output_checker_array == 0 || 
	device_input_array == 0 || device_output_array == 0)
  {
    printf("couldn't allocate memory\n");
    return 1;
  }


  // generate random input string
  unsigned int shift_amount = rand();
  
  for(int i=0;i< num_elements;i++)
  {
    host_input_array[i] = (unsigned int)rand(); 
  }
  
  // do copies to and from gpu once to get rid of timing weirdness
  // on first time accesses due to driver
  hipMemcpy(device_input_array, host_input_array, num_bytes, hipMemcpyHostToDevice);
  hipMemcpy(host_output_array, device_output_array, num_bytes, hipMemcpyDeviceToHost);

  start_timer(&timer);
  // copy input to GPU
  hipMemcpy(device_input_array, host_input_array, num_bytes, hipMemcpyHostToDevice);
  check_launch("copy to gpu");
  stop_timer(&timer,"copy to gpu");
  
  // choose a number of threads per block
  // we use 512 threads here
  int block_size = 512;

  int grid_size = (num_elements + block_size - 1) / block_size;

  start_timer(&timer);
  // launch kernel
  shift_cypher<<<grid_size,block_size>>>(device_input_array, device_output_array, shift_amount, alphabet_max, num_elements);
  check_launch("gpu shift cypher");
  stop_timer(&timer,"gpu shift cypher");

  start_timer(&timer);
  // download and inspect the result on the host:
  hipMemcpy(host_output_array, device_output_array, num_bytes, hipMemcpyDeviceToHost);
  check_launch("copy from gpu");
  stop_timer(&timer,"copy from gpu");
  
  start_timer(&timer);
  // generate reference output
  host_shift_cypher(host_input_array, host_output_checker_array, shift_amount, alphabet_max, num_elements);
  stop_timer(&timer,"host shift cypher");
  
  // check CUDA output versus reference output
  int error = 0;
  for(int i=0;i<num_elements;i++)
  {
    if(host_output_array[i] != host_output_checker_array[i]) 
    { 
      error = 1;
    }
  }
  
  if(error)
  {
    printf("Output of CUDA version and normal version didn't match! \n");
  }
  else
  {
    printf("Worked! CUDA and reference output match. \n");
  }
 
  // deallocate memory
  free(host_input_array);
  free(host_output_array);
  free(host_output_checker_array);
  hipFree(device_input_array);
  hipFree(device_output_array);
}

