#include "hip/hip_runtime.h"
/* This is machine problem 1, part 2, force evaluation
 *
 * The problem is to take two sets of charged particles, 
 * where each particle has a position and a charge associated with itself,
 * and calculate the force between specific pairs of particles. 
 * An index array holds the information which particle in set B should be
 * paired with which particle in set A.
 * SUBMISSION GUIDELINES:
 * You should submit two files, called mp1-part2-solution-kernel.cu and mp1-part2-solution-host.cu
 * which contain your version of the force_eval and host_charged_particles functions.
 */


#include <stdlib.h>
#include <stdio.h>

#include "mp1-util.h"
#define EPSILON 0.00001f

// amount of floating point numbers between answer and computed value 
// for the answer to be taken correctly. 2's complement magick.
const int maxUlps = 1000;

event_pair timer;
  
float4 force_calc(float4 A, float4 B) 
{
  float x = B.x - A.x;
  float y = B.y - A.y;
  float z = B.z - A.z;
  float rsq = x*x + y*y + z*z;
  // avoid divide by zero
  if(rsq < EPSILON)
  {
    rsq += EPSILON;
  }
  float r = sqrt(rsq);
  float f = A.w * B.w / rsq;
  float inv_r = 1.0f / r;
  float4 fv = make_float4(x*inv_r,y*inv_r,z*inv_r,f);
  return fv;
}
 
void host_force_eval(float4 *set_A, float4 *set_B, int * indices, float4 *force_vectors, int array_length)
{
  for(int i=0;i<array_length;i++)
  {
    if(indices[i] < array_length && indices[i] >= 0)
    {
      force_vectors[i] = force_calc(set_A[i],set_B[indices[i]]);
    }
    else
    {
      force_vectors[i] = make_float4(0.0,0.0,0.0,0.0);
    }
  }
}


__global__ void force_eval(float4 *set_A, float4 *set_B, int * indices, float4 *force_vectors, int array_length)
{
  // TODO your code here ...
}



void host_charged_particles(float4 *h_set_A, float4 *h_set_B, int *h_indices, float4 *h_force_vectors, int num_elements)
{ 
  // TODO your code here ...
  
  start_timer(&timer);
  // launch kernel
  
  // the actual kernel launch should go here, so that the time it took is measured 

  check_launch("gpu force eval");
  stop_timer(&timer,"gpu force eval");
  
  // TODO more code here...
}


int main(void)
{
  // create arrays of 4M elements
  int num_elements =  1 << 22;

  // pointers to host & device arrays
  float4 *h_set_A = 0;
  float4 *h_set_B = 0;
  int *h_indices = 0;
  float4 *h_force_vectors = 0;
  float4 *h_force_vectors_checker = 0;
  
   // initialize
  srand(time(NULL)); 
  
  // malloc host array
  h_set_A = (float4*)malloc(num_elements * sizeof(float4));
  h_set_B = (float4*)malloc(num_elements * sizeof(float4));
  h_indices = (int*)malloc(num_elements * sizeof(int));
  h_force_vectors = (float4*)malloc(num_elements * sizeof(float4));
  h_force_vectors_checker = (float4*)malloc(num_elements * sizeof(float4));
  
  // if either memory allocation failed, report an error message
  if(h_set_A == 0 || h_set_B == 0 || h_force_vectors == 0 || h_indices == 0 || h_force_vectors_checker == 0)
  {
    printf("couldn't allocate memory\n");
    exit(1);
  }

  // generate random input
  for(int i=0;i< num_elements;i++)
  {
    h_set_A[i] = make_float4(rand(),rand(),rand(),rand()); 
    h_set_B[i] = make_float4(rand(),rand(),rand(),rand());

    // some indices will be invalid
    h_indices[i] = rand() % (num_elements + 2);
  }
  
  start_timer(&timer);
  // generate reference output
  host_force_eval(h_set_A, h_set_B, h_indices, h_force_vectors_checker, num_elements);
  
  check_launch("host force eval");
  stop_timer(&timer,"host force eval");
  
  // the results of the calculation need to end up in h_force_vectors;
  host_charged_particles(h_set_A, h_set_B, h_indices, h_force_vectors, num_elements);
  
  // check CUDA output versus reference output
  int error = 0;
  
  for(int i=0;i<num_elements;i++)
  {
    float4 v = h_force_vectors[i];
    float4 vc = h_force_vectors_checker[i];

    if( !AlmostEqual2sComplement(v.x,vc.x,maxUlps) ||
    	!AlmostEqual2sComplement(v.y,vc.y,maxUlps) ||
    	!AlmostEqual2sComplement(v.z,vc.z,maxUlps) ||
    	!AlmostEqual2sComplement(v.w,vc.w,maxUlps)) 
    { 
      error = 1;
    }
  }
  printf("\n");
  
  if(error)
  {
    printf("Output of CUDA version and normal version didn't match! \n");
  }
  else
  {
    printf("Worked! CUDA and reference output match. \n");
  }
 
  // deallocate memory
  free(h_set_A);
  free(h_set_B);
  free(h_indices);
  free(h_force_vectors);
  free(h_force_vectors_checker);
}

